#include "hip/hip_runtime.h"
#include "vars.h"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void P2Mkernel(hipDoubleComplex *C, hipDoubleComplex *A, hipDoubleComplex *B, int M, int N, int L){
  extern __shared__ hipDoubleComplex temp[];
  int tile = blockDim.x;
  int row = blockIdx.y*tile+threadIdx.y;
  int col = blockIdx.x*tile+threadIdx.x;
  int numtile = N/tile;
  if(N%tile)numtile++;
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  int index = threadIdx.y*tile+threadIdx.x;
  int block = tile*tile;
  int start = 0;
  for(int k = 0; k < numtile; k++){
    if(row < M && start+threadIdx.x < N)
        temp[index] = A[row*N+start+threadIdx.x];
    if(col < L && start+threadIdx.y < N)
      temp[block+index] = B[(start+threadIdx.y)*L+col];
    __syncthreads();
    if(col < L && row < M)
      for(int i = 0; i < tile && start + i < N; i++)
        reduce = hipCadd(reduce,hipCmul(temp[threadIdx.y*tile+i],temp[block+i*tile+threadIdx.x]));
    __syncthreads();
    start = start+tile;
  }
  if(col < L && row < M)C[row*L+col] = reduce;
}

__global__ void L2Pkernel(hipDoubleComplex *C, hipDoubleComplex *A, hipDoubleComplex *B, int M, int N, int L){
  extern __shared__ hipDoubleComplex temp[];
  int tile = blockDim.x;
  int row = blockIdx.y*tile+threadIdx.y;
  int col = blockIdx.x*tile+threadIdx.x;
  int numtile = N/tile;
  if(N%tile)numtile++;
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  int index = threadIdx.y*tile+threadIdx.x;
  int block = tile*tile;
  int start = 0;
  for(int k = 0; k < numtile; k++){
    if(row < M && start+threadIdx.x < N)
      temp[index] = A[row*N+start+threadIdx.x];
    if(col < L && start+threadIdx.y < N)
      temp[block+index] = B[(start+threadIdx.y)*L+col];
    __syncthreads();
    if(col < L && row < M)
      for(int i = 0; i < tile && start + i < N; i++)
        reduce = hipCadd(reduce,hipCmul(temp[threadIdx.y*tile+i],temp[block+i*tile+threadIdx.x]));
    __syncthreads();
    start = start+tile;
  }
  if(col < L && row < M)C[row*L+col] = hipCadd(C[row*L+col],reduce);
}
__global__ void L2Pkernelh(hipDoubleComplex *C, hipDoubleComplex *A, hipDoubleComplex *B, int M, int N, int L){
  extern __shared__ hipDoubleComplex temp[];
  int tile = blockDim.x;
  int row = blockIdx.y*tile+threadIdx.y;
  int col = blockIdx.x*tile+threadIdx.x;
  int numtile = N/tile;
  if(N%tile)numtile++;
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  int index = threadIdx.y*tile+threadIdx.x;
  int block = tile*tile;
  int start = 0;
  for(int k = 0; k < numtile; k++){
    if(row < M && start+threadIdx.x < N)
      temp[index] = A[row*N+start+threadIdx.x];
    if(col < L && start+threadIdx.y < N)
      temp[block+index] = B[(start+threadIdx.y)*L+col];
    __syncthreads();
    if(col < L && row < M)
      for(int i = 0; i < tile && start + i < N; i++)
        reduce = hipCadd(reduce,hipCmul(temp[threadIdx.y*tile+i],temp[block+i*tile+threadIdx.x]));
    __syncthreads();
    start = start+tile;
  }
  if(col < L && row < M)C[row*L+col] = reduce;
}

__global__ void M2Mkernel(hipDoubleComplex *agg, hipDoubleComplex *aggl, int sampl, double *interp, int *intind, int ninter, hipDoubleComplex *shift){
  extern __shared__ hipDoubleComplex temp[];
  int clusm = blockIdx.y*gridDim.x+blockIdx.x;
  int start = clusm*4;
  if(threadIdx.x<sampl)
    for(int cn = 0; cn < 4; cn++)
      temp[cn*sampl+threadIdx.x] = aggl[(start+cn)*sampl+threadIdx.x];
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  hipDoubleComplex reg[4] = {0, 0, 0, 0};
  hipDoubleComplex intval;
  int indval = intind[threadIdx.x];
  __syncthreads();
  for(int k = 0; k < ninter; k++){
    intval = make_hipDoubleComplex(interp[k*blockDim.x+threadIdx.x],0);
    for(int m = 0; m < 4; m++)
      reg[m] = hipCadd(reg[m],hipCmul(intval,temp[m*sampl+(indval+k)%sampl]));
  }
  for(int m = 0; m < 4; m++)
    reduce = hipCadd(reduce,hipCmul(reg[m],shift[m*blockDim.x+threadIdx.x]));
  agg[clusm*blockDim.x+threadIdx.x] = reduce;
}
__global__ void M2Mkernel_output(hipDoubleComplex *agg, hipDoubleComplex *aggl, int samp, int sampl, int numclus, double *interp, int *intind, int ninter, hipDoubleComplex *shift){
  extern __shared__ hipDoubleComplex temp[];
  hipDoubleComplex *shifttemp = &temp[blockDim.x*ninter];
  int *tempind = (int*)&shifttemp[4*blockDim.x];
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  hipDoubleComplex reg[4] = {0,0,0,0};
  int index = threadIdx.y*blockDim.x+threadIdx.x;
  int size = blockDim.y*blockDim.x;
  int start = blockIdx.x*blockDim.x*ninter;
  int ind = threadIdx.y;
  if(col < samp){
    while(ind < ninter){
      temp[ind*blockDim.x+threadIdx.x] = make_hipDoubleComplex(interp[ind*samp+col],0);
      ind = ind + blockDim.y;
    }
    if(threadIdx.y < 4)shifttemp[index] = shift[threadIdx.y*samp+col];
    if(threadIdx.y == 0)tempind[threadIdx.x] = intind[col];
  }
  __syncthreads();
  if(col < samp && row < numclus){
    for(int k = 0; k < ninter; k++){
      ind = (tempind[threadIdx.x]+k)%sampl;
      hipDoubleComplex coeff = temp[k*blockDim.x+threadIdx.x];
      for(int m = 0; m < 4; m++)
        reg[m] = hipCadd(reg[m],hipCmul(coeff,aggl[(4*row+m)*sampl+ind]));
    }
    for(int m = 0; m < 4; m++)
      reduce = hipCadd(reduce,hipCmul(reg[m],shifttemp[m*blockDim.x+threadIdx.x]));
    agg[row*samp+col] = reduce;
  }
}
__global__ void L2Lkernel_output(hipDoubleComplex *agg, hipDoubleComplex *aggl, int samp, int sampl, int numclus, double *interp, int *intind, int ninter, hipDoubleComplex *shift){
  extern __shared__ hipDoubleComplex temp[];
  int *tempind = (int*)&temp[ninter*blockDim.x];
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  hipDoubleComplex reg[4] = {0,0,0,0};
  int index = threadIdx.y*blockDim.x+threadIdx.x;
  int size = blockDim.y*blockDim.x;
  int start = blockIdx.x*blockDim.x*ninter;
  int ind = threadIdx.y;
  if(col < sampl){
    while(ind < ninter){
      temp[ind*blockDim.x+threadIdx.x] = make_hipDoubleComplex(interp[ind*sampl+col],0);
      ind = ind + blockDim.y;
    }
  }
  if(threadIdx.y==0)tempind[threadIdx.x] = intind[col];
  __syncthreads();
  if(col < sampl && row < numclus){
    for(int k = 0; k < ninter; k++){
      int ind = (tempind[threadIdx.x]+k)%samp;
      hipDoubleComplex coeff = temp[k*blockDim.x+threadIdx.x];
      hipDoubleComplex sample = hipCmul(coeff,agg[row*samp+ind]);
      for(int m = 0; m < 4; m++)
        reg[m] = hipCadd(reg[m],hipCmul(sample,shift[m*samp+ind]));
    }
    for(int m = 0; m < 4; m++)
      aggl[(4*row+m)*sampl+col] = hipCadd(aggl[(4*row+m)*sampl+col],reg[m]);
  }
}
__global__ void L2Lkernelh_output(hipDoubleComplex *agg, hipDoubleComplex *aggl, int samp, int sampl, int numclus, double *interp, int *intind, int ninter, hipDoubleComplex *shift){
  extern __shared__ hipDoubleComplex temp[];
  int *tempind = (int*)&temp[ninter*blockDim.x];
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  hipDoubleComplex reg[4] = {0,0,0,0};
  int index = threadIdx.y*blockDim.x+threadIdx.x;
  int size = blockDim.y*blockDim.x;
  int start = blockIdx.x*blockDim.x*ninter;
  int ind = threadIdx.y;
  if(col < sampl){
    while(ind < ninter){
      temp[ind*blockDim.x+threadIdx.x] = make_hipDoubleComplex(interp[ind*sampl+col],0);
      ind = ind + blockDim.y;
    }
  }
  if(threadIdx.y==0)tempind[threadIdx.x] = intind[col];
  __syncthreads();
  if(col < sampl && row < numclus){
    for(int k = 0; k < ninter; k++){
      int ind = (tempind[threadIdx.x]+k)%samp;
      hipDoubleComplex coeff = temp[k*blockDim.x+threadIdx.x];
      hipDoubleComplex sample = hipCmul(coeff,agg[row*samp+ind]);
      for(int m = 0; m < 4; m++)
        reg[m] = hipCadd(reg[m],hipCmul(sample,shift[m*samp+ind]));
    }
    for(int m = 0; m < 4; m++)
      aggl[(4*row+m)*sampl+col] = reg[m];
  }
}
__global__ void L2Lkernel(hipDoubleComplex *agg, hipDoubleComplex *aggl, int sampl, double *interp, int *intind, int ninter, hipDoubleComplex *shift){
  extern __shared__ hipDoubleComplex temp[];
  int clusn = blockIdx.y*gridDim.x+blockIdx.x;
  hipDoubleComplex samp = agg[clusn*blockDim.x+threadIdx.x];
  for(int cn = 0; cn < 4; cn++)
    temp[cn*blockDim.x+threadIdx.x] = hipCmul(samp,shift[cn*blockDim.x+threadIdx.x]);
  hipDoubleComplex reg[4] = {0,0,0,0};
  hipDoubleComplex intval;
  int indval = intind[threadIdx.x];
  __syncthreads();
  if(threadIdx.x < sampl){
    for(int k = 0; k < ninter; k++){
      intval = make_hipDoubleComplex(interp[k*sampl+threadIdx.x],0);
      int ind = (indval+k)%blockDim.x;
      for(int m = 0; m < 4; m++)
        reg[m] = hipCadd(reg[m],hipCmul(intval,temp[m*blockDim.x+ind]));
    }
    for(int m = 0; m < 4; m++)
      aggl[(clusn*4+m)*sampl+threadIdx.x] = hipCadd(aggl[(clusn*4+m)*sampl+threadIdx.x],reg[m]);
  }
}
__global__ void L2Lkernelh(hipDoubleComplex *agg, hipDoubleComplex *aggl, int sampl, double *interp, int *intind, int ninter, hipDoubleComplex *shift){
  extern __shared__ hipDoubleComplex temp[];
  int clusn = blockIdx.y*gridDim.x+blockIdx.x;
  hipDoubleComplex samp = agg[clusn*blockDim.x+threadIdx.x];
  for(int cn = 0; cn < 4; cn++)
    temp[cn*blockDim.x+threadIdx.x] = hipCmul(samp,shift[cn*blockDim.x+threadIdx.x]);
  hipDoubleComplex reg[4] = {0,0,0,0};
  hipDoubleComplex intval;
  int indval = intind[threadIdx.x];
  __syncthreads();
  if(threadIdx.x < sampl){
    for(int k = 0; k < ninter; k++){
      intval = make_hipDoubleComplex(interp[k*sampl+threadIdx.x],0);
      int ind = (indval+k)%blockDim.x;
      for(int m = 0; m < 4; m++)
        reg[m] = hipCadd(reg[m],hipCmul(intval,temp[m*blockDim.x+ind]));
    }
    for(int m = 0; m < 4; m++)
      aggl[(clusn*4+m)*sampl+threadIdx.x] = reg[m];
  }
}
__global__ void M2Lkernel_output(hipDoubleComplex *loc, hipDoubleComplex *agg, int samp, int numclus, int *far,int *traid, hipDoubleComplex *trans){
  extern __shared__ int tempfar[];
  int *tempid = &tempfar[blockDim.y*27];
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  int index = threadIdx.y*blockDim.x+threadIdx.x;
  int size = blockDim.y*blockDim.x;
  int start = blockIdx.y*blockDim.y*27;
  int ind = index;
  while(ind < blockDim.y*27){
    tempfar[ind] = far[start+ind];
    tempid[ind] = traid[start+ind];
    ind = ind + size;
  }
  __syncthreads();
  if(col < samp && row < numclus){
    for(int cn = 0; cn < 27; cn++){
      int clusn = tempfar[threadIdx.y*27+cn];
      if(clusn != -1){
        int indmulti = clusn*samp;
        int index = tempid[threadIdx.y*27+cn]*samp;
        reduce = hipCadd(reduce,hipCmul(trans[index+col],agg[indmulti+col]));
      }
    }
    loc[row*samp+col] = reduce;
  }
}
/*__global__ void P2Pkernel(hipDoubleComplex *r, hipDoubleComplex *x, int *clusnear, hipDoubleComplex *near){
  extern __shared__ hipDoubleComplex temp[];
  int box = blockDim.x;
  int boxx = box*box;
  int *clutemp = (int*)&temp[boxx];
  int t = threadIdx.y*box+threadIdx.x;
  int clusm = blockIdx.y*gridDim.x+blockIdx.x;
  if(t < 9)clutemp[t] = clusnear[clusm*9+t];
  __syncthreads();
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  int clusn, indboxbase;
  for(int cn = 0; cn < 9; cn++){
    clusn = clutemp[cn];
    if(clusn != -1){
      __syncthreads();
      temp[t] = x[clusn*boxx+t];
      indboxbase = cn*boxx*boxx;
      __syncthreads();
      for(int n = 0; n < boxx; n++)
        reduce = hipCadd(reduce,hipCmul(temp[n],near[indboxbase+n*boxx+t]));
    }
  }
  r[clusm*boxx+t] = reduce;
}*/
//LARGE BLOCK
__global__ void P2Pkernel(hipDoubleComplex *r, hipDoubleComplex *x, int *clusnear, hipDoubleComplex *near){
  extern __shared__ hipDoubleComplex temp[];
  hipDoubleComplex *neartemp = &temp[blockDim.x];
  int boxx = blockDim.x/4;
  int t = threadIdx.x%boxx;
  int clusl = threadIdx.x/boxx;
  int clusm = blockIdx.x*4+clusl;
  int clusn,indboxbase;
  int *nearlist = (int*)&neartemp[blockDim.x];
  hipDoubleComplex reduce = make_hipDoubleComplex(0,0);
  if(t<9)nearlist[clusl*9+t] = clusnear[clusm*9+t];
  __syncthreads();
  for(int cn = 0; cn < 9; cn++){
    clusn = nearlist[clusl*9+cn];
    indboxbase = cn*boxx*boxx;
    __syncthreads();
    if(clusn!=-1)temp[threadIdx.x] = x[clusn*boxx+t];
    for(int n = 0; n < boxx; n++){
      if(n%4==0){
        __syncthreads();
        neartemp[threadIdx.x]=near[indboxbase+n*boxx+threadIdx.x];
        __syncthreads();
      }
      if(clusn != -1)reduce = hipCadd(reduce,hipCmul(neartemp[(n%4)*boxx+t],temp[clusl*boxx+n]));
    }
  }
  r[clusm*boxx+t] = reduce;
}
__global__ void locate(hipDoubleComplex *sendbuff, int *sendmap, hipDoubleComplex *aggmulti, int numsamp){
  int loop = 0;
  for(loop = 0; loop < numsamp/blockDim.x; loop++)
    sendbuff[blockIdx.x*numsamp+loop*blockDim.x+threadIdx.x] = aggmulti[sendmap[blockIdx.x]*numsamp+loop*blockDim.x+threadIdx.x];
  if(threadIdx.x < numsamp%blockDim.x)
    sendbuff[blockIdx.x*numsamp+loop*blockDim.x+threadIdx.x] = aggmulti[sendmap[blockIdx.x]*numsamp+loop*blockDim.x+threadIdx.x];
}
__global__ void relocate(hipDoubleComplex *aggmulti, int *recvmap, hipDoubleComplex *recvbuff, int numsamp){
  int loop = 0;
  for(loop = 0; loop < numsamp/blockDim.x; loop++)
    aggmulti[recvmap[blockIdx.x]*numsamp+loop*blockDim.x+threadIdx.x] = recvbuff[blockIdx.x*numsamp+loop*blockDim.x+threadIdx.x];
  if(threadIdx.x < numsamp%blockDim.x)
    aggmulti[recvmap[blockIdx.x]*numsamp+loop*blockDim.x+threadIdx.x] = recvbuff[blockIdx.x*numsamp+loop*blockDim.x+threadIdx.x];
}
__global__ void norm2partc(double *part, hipDoubleComplex *a){
  extern __shared__ double tempd[];
  int start = 2*blockIdx.x*blockDim.x;
  hipDoubleComplex var;
  var = a[start+threadIdx.x];
  tempd[threadIdx.x] = hipCreal(var)*hipCreal(var)+hipCimag(var)*hipCimag(var);
  var = a[start+blockDim.x+threadIdx.x];
  tempd[blockDim.x+threadIdx.x] = hipCreal(var)*hipCreal(var)+hipCimag(var)*hipCimag(var);
  for(int stride = blockDim.x; stride >= 1; stride >>= 1){
    __syncthreads();
    if(threadIdx.x < stride)
      tempd[threadIdx.x]=tempd[threadIdx.x]+tempd[threadIdx.x+stride];
  }
  if(threadIdx.x==0)
    part[blockIdx.x] = tempd[0];
}
__global__ void norm2partd(double *part){
  extern __shared__ double tempd[];
  int start = 2*blockIdx.x*blockDim.x;
  tempd[threadIdx.x] = part[start+threadIdx.x];
  tempd[blockDim.x+threadIdx.x] = part[start+blockDim.x+threadIdx.x];
  for(int stride = blockDim.x; stride >= 1; stride >>= 1){
    __syncthreads();
    if(threadIdx.x < stride)
      tempd[threadIdx.x]=tempd[threadIdx.x]+tempd[threadIdx.x+stride];
  }
  if(threadIdx.x==0)
    part[blockIdx.x] = tempd[0];
}
__global__ void innerpartc(hipDoubleComplex *part, hipDoubleComplex *a, hipDoubleComplex *b){
  extern __shared__ hipDoubleComplex temp[];
  int start = 2*blockIdx.x*blockDim.x;
  hipDoubleComplex var1, var2;
  var1 = a[start+threadIdx.x];
  var2 = b[start+threadIdx.x];
  temp[threadIdx.x] = hipCmul(hipConj(var1),var2);
  var1 = a[start+blockDim.x+threadIdx.x];
  var2 = b[start+blockDim.x+threadIdx.x];
  temp[blockDim.x+threadIdx.x] = hipCmul(hipConj(var1),var2);
  for(int stride = blockDim.x; stride >= 1; stride >>= 1){
    __syncthreads();
    if(threadIdx.x < stride)
      temp[threadIdx.x]=hipCadd(temp[threadIdx.x],temp[threadIdx.x+stride]);
  }
  if(threadIdx.x==0)
    part[blockIdx.x] = temp[0];
}
__global__ void innerpartd(hipDoubleComplex *part){
  extern __shared__ hipDoubleComplex temp[];
  int start = 2*blockIdx.x*blockDim.x;
  temp[threadIdx.x] = part[start+threadIdx.x];
  temp[blockDim.x+threadIdx.x] = part[start+blockDim.x+threadIdx.x];
  for(int stride = blockDim.x; stride >= 1; stride >>= 1){
    __syncthreads();
    if(threadIdx.x < stride)
      temp[threadIdx.x]=hipCadd(temp[threadIdx.x],temp[threadIdx.x+stride]);
  }
  if(threadIdx.x==0)
    part[blockIdx.x] = temp[0];
}
__global__ void prep(hipDoubleComplex *buff, hipDoubleComplex *x, hipDoubleComplex *o){
  int t = blockIdx.x*blockDim.x+threadIdx.x;
  buff[t] = hipCmul(x[t],o[t]);
}
__global__ void prep(hipDoubleComplex *buff, hipDoubleComplex *x){
  int t = blockIdx.x*blockDim.x+threadIdx.x;
  buff[t] = hipConj(x[t]);
}
__global__ void post(hipDoubleComplex *r_d, hipDoubleComplex *x_d){
  int t = blockIdx.x*blockDim.x+threadIdx.x;
  r_d[t] = hipCsub(x_d[t],r_d[t]);
}
__global__ void post(hipDoubleComplex *r, hipDoubleComplex *x, hipDoubleComplex *o){
  int t = blockIdx.x*blockDim.x+threadIdx.x;
  r[t] = hipCsub(x[t],hipConj(hipCmul(o[t],r[t])));
}
__global__ void saxp(hipDoubleComplex *a,hipDoubleComplex *b,hipDoubleComplex *c,hipDoubleComplex alpha){
  int t = blockIdx.x*blockDim.x+threadIdx.x;
  c[t] = hipCadd(a[t],hipCmul(alpha,b[t]));
}
